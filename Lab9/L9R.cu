//Compile the code and run it with a parameter
//For example
//     ./L9 1024
//means the size of the matrices is 1024

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <string.h>
#define LENB 16

__global__ void My_MMult(double *A, double *B, double *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        double sum = C[row * n + col];
        for (int i = 0; i < k; ++i) 
            sum += A[row * k + i] * B[i * n + col];
        C[row * n + col] = sum;
    }
}


double TIME() {
    struct timeval t;
    gettimeofday(&t, NULL);
    return ((double)t.tv_sec + (double)t.tv_usec * 1e-6);
}

int main(int argc, char**argv) {
    int m=1, n=1, k=1;
    m = atoi(argv[1]);
    n = atoi(argv[1]);
    k = atoi(argv[1]);
    double *A, *B, *C, *a, *b, *c;
    A = (double *)malloc(m * k * sizeof(double));
    B = (double *)malloc(k * n * sizeof(double));
    C = (double *)malloc(m * n * sizeof(double));
    hipMalloc((void **)&a, m * k * sizeof(double));
    hipMalloc((void **)&b, k * n * sizeof(double));
    hipMalloc((void **)&c, m * n * sizeof(double));
    for (int i = 0; i < m; i++)
        for (int j = 0; j < k;j++)
            A[i*k+j]=1;
    for (int i = 0; i < k; i++)
        for (int j = 0; j < n;j++)
            B[i*n+j]=1;
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n;j++)
            C[i*n+j]=1;
    hipMemcpy(a, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b, B, k * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c, C, m * n * sizeof(double), hipMemcpyHostToDevice);
    dim3 blockSize(LENB, LENB);
    dim3 gridSize((n + LENB - 1) / LENB, (m + LENB - 1) / LENB);

    double st= TIME();
    My_MMult<<<gridSize, blockSize>>>(a, b, c, m, n, k);
    hipDeviceSynchronize();
    double et = TIME();

    double gflops = 2.0 * m * k * n * 1e-9;
    gflops /= et - st;

    printf("%d ",m);
    printf("%le\n", gflops);
    
    hipMemcpy(C, c, m * n * sizeof(double), hipMemcpyDeviceToHost);
/*
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n;j++)
            printf("%.0lf", C[i * n + j]);
        printf("\n");
    }
*/
    free(A);free(B);free(C);
    hipFree(a);hipFree(b);hipFree(c);
    return 0;
}
